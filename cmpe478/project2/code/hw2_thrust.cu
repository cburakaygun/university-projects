#include "hip/hip_runtime.h"
#include <time.h>
#include <fstream>
#include <iostream>
#include <unordered_map>
#include <vector>

#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/inner_product.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/reduce.h>
#include <thrust/transform.h>


#define NODE_NUM 1850065  // Number of nodes (vertices) of the graph.
#define EDGE_NUM 16741171 // Number of edges of the graph.

#define ALPHA 0.2
#define EPSILON 0.000001


using namespace std;


// Absolute value of the difference of the given arguments
struct differenceAbs_functor
{
    __host__ __device__
    double operator()(const double& x, const double& y) const {
        return abs(x - y);
    }
};



// Reads Erdös Web Graph file and creates related P matrix in CSR format.
void createCSRArrays(char *graphFilePath, vector<int> &rowBegin, vector<double> &values, vector<int> &columnIndices) {

    int i; // Loop variable
    string node1, node2; // Represents 2 adjacent nodes in the graph.

    int IN_NODE_NUM; // Number of nodes with incoming edges.

    vector<pair<string, string>> graphVector(EDGE_NUM); // Stores graph.txt file.
    unordered_map<string, int> nodeIndexMap;  // Maps node IDs to a row/column index of P matrix.

    // graphFilePath is the path of graph.txt file.
    ifstream graphFile(graphFilePath);

    printf("Reading %s\t\t\t\t\t\t\t... ", graphFilePath);
    fflush(stdout);

    int graphVector_i = 0;
    // Iterates over graph.txt file.
    while (graphFile >> node1 >> node2) { // There is a directed edge from node 1 to node 2.
        graphVector[graphVector_i++] = make_pair(node1, node2);

        if (nodeIndexMap.find(node2) == nodeIndexMap.end()) { // If node 2 is not in `nodeIndexMap` ...
            nodeIndexMap.emplace(node2, nodeIndexMap.size()); // ... inserts it with the next index.
        }
    }

    graphFile.close();
    printf("DONE\n");

    IN_NODE_NUM = nodeIndexMap.size(); // The WHILE-LOOP above inserts only the nodes with incoming edges to `nodeIndexMap`.

    printf("Constructing CSR-format related arrays\t\t\t\t\t... ");
    fflush(stdout);

    vector<int> outDegrees(NODE_NUM, 0); // Stores the out degree of each node.

    auto graphVectorItr = graphVector.begin();

    int indexOfNextRow = 0;  // Index of (next) row in values (and columnIndices) arrays.
    int rowBegin_i = 0;
    int columnIndices_i = 0;

    // First IN_NODE_NUM indices are assigned to the nodes with incoming edges.
    for (int nodeIndex = 0; nodeIndex < IN_NODE_NUM; nodeIndex++) {
        rowBegin[rowBegin_i++] = indexOfNextRow;

        while (graphVectorItr != graphVector.end()) {
            node1 = graphVectorItr->first;
            node2 = graphVectorItr->second;

            if (nodeIndexMap.find(node2)->second != nodeIndex) {
                break;  // If index of `node2` is not equal to `nodeIndex`, breaks the WHILE-LOOP.
            }

            if (nodeIndexMap.find(node1) == nodeIndexMap.end()) { // If node 1 is not in `nodeIndexMap` ...
                nodeIndexMap.emplace(node1, nodeIndexMap.size()); // ... inserts it with the next index.
            }

            int node1Index = nodeIndexMap.find(node1)->second;
            outDegrees[node1Index]++; // There is a directed edge from node 1 to node 2.
            columnIndices[columnIndices_i++] = node1Index;
            indexOfNextRow++;
            graphVectorItr++;
        }
    }

    // Inserts row indices for all-zero rows of P matrix.
    for (i = rowBegin_i; i < NODE_NUM + 1; i++) {
        rowBegin[i] = EDGE_NUM;
    }

    for (i = 0; i < EDGE_NUM; i++) {
        int nodeIndex = columnIndices[i];
        double value = 1.0 / outDegrees[nodeIndex];
        values[i] = value;
    }

    printf("DONE\n");  // CSR-format related arrays are constructed.

}


int main(int argc, char *argv[]) {

    vector<int> rowBegin(NODE_NUM+1); // Row indices array of the CSR format of P matrix.
    vector<double> values(EDGE_NUM); // Nonzero values of P matrix.
    vector<int> columnIndices(EDGE_NUM); // Column indices array of the CSR format of P matrix.

    createCSRArrays(argv[1], rowBegin, values, columnIndices);

    printf("\n");

    thrust::device_vector<double > values_D(values); // Copies values array from host to device

    thrust::device_vector<double> r1_D(NODE_NUM, 1); // r^(t)
    thrust::device_vector<double> r2_D(NODE_NUM, 0); // r^(t+1)
    thrust::device_vector<double> difference_D(NODE_NUM, 0);  // ith element = |r_i^(t+1) - r_i^(t)|

    int iterationCount = 0;

    auto clockStart = clock();

    while(true) {
        iterationCount++;

        // r2_D = P * r1_D
        for (int i=0; i<NODE_NUM; i++){
            int x = rowBegin[i];   // Begin of row-i
            int y = rowBegin[i+1]; // End of row-i

            auto permIter = thrust::make_permutation_iterator(r1_D.begin(), columnIndices.begin()+x);

            r2_D[i] = ALPHA * thrust::inner_product(values_D.begin()+x, values_D.begin()+y, permIter, 0.0) + (1-ALPHA);
        }

        // difference_D[i] = | r2_D[i] - r1_D[i] |
        thrust::transform(r2_D.begin(), r2_D.end(), r1_D.begin(), difference_D.begin(), differenceAbs_functor());

        double result = thrust::reduce(difference_D.begin(), difference_D.end());
        printf("Difference after %02d iterations: %f\n", iterationCount, result);

        if (result <= EPSILON) {
            break;
        } else {
            thrust::copy(r2_D.begin(), r2_D.end(), r1_D.begin());
        }
    }

    float elapsedTime = (float)(clock() - clockStart) / CLOCKS_PER_SEC;
    printf("\nTime Elapsed for Ranking Algorithm: %f sec\n", elapsedTime);

    return 0;
}
